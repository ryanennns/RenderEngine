#include "hip/hip_runtime.h"
#include "../include/Coordinates.h"
#include "stdio.h"
#include "hip/hip_runtime.h"

// CUDA kernel that operates on primitives and outputs coordinates
__global__ void generateCoordinatesKernel(
    const int width,
    const int height,
    const int x,
    const double aspectRatio,
    Coordinates *output
)
{
    const unsigned int idx = threadIdx.x;
    if (idx < width) {
        double normalizedX = (idx + 0.5) / width;
        double normalizedY = (x + 0.5) / height;

        normalizedX = (2 * normalizedX) - 1;
        normalizedY = (2 * normalizedY) - 1;

        normalizedX *= aspectRatio;

        output[idx] = {normalizedX, normalizedY};
    }
}

extern "C" void generateCoordinatesOnGPU(
    const int width,
    const int height,
    const int x,
    const double aspectRatio,
    Coordinates *output
)
{
    Coordinates *d_output = nullptr;
    const size_t size = height * sizeof(Coordinates);

    hipMalloc((void **) &d_output, size);

    generateCoordinatesKernel<<<1, height>>>(width, height, x, aspectRatio, d_output);
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_output);
}
