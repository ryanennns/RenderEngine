#include "hip/hip_runtime.h"
#include "../include/Structs.h"
#include "../include/helpers.h"
#include <cstdio>
#include "hip/hip_runtime.h"

__global__ void generateCoordinatesKernel(
    const int width,
    const int height,
    const int x,
    const double aspectRatio,
    Coordinates *output
)
{
    const unsigned int idx = threadIdx.x;
    if (idx < width) {
        double normalizedX = (idx + 0.5) / width;
        double normalizedY = (x + 0.5) / height;

        normalizedX = (2 * normalizedX) - 1;
        normalizedY = (2 * normalizedY) - 1;

        normalizedX *= aspectRatio;

        output[idx] = {normalizedX, normalizedY};
    }
}

__global__ void intersectionKernel(
    const Line *lines,
    const Landscape *landscape,
    const int width,
    const int height,
    LineTriangleIntersection *objectIntersections
)
{
    const unsigned int idx = blockIdx.x;
    const unsigned int idy = threadIdx.x;

    if (idx > width || idy > height) {
        return;
    }

    unsigned int index = idy * width + idx;
    LineTriangleIntersection intersect = lineIntersectsLandscape(lines[index], *landscape);

    objectIntersections[index] = intersect;
}

__host__ void copyLandscapeToGPU(
    const Landscape landscape,
    Landscape **d_landscape
)
{
    hipMalloc((void **) &d_landscape, sizeof(Landscape));

    Object *d_objects = nullptr;
    hipMalloc((void **) &d_objects, landscape.size * sizeof(Object));

    for (int i = 0; i < landscape.size; i++) {
        const Object object = landscape.objects[i];

        Triangle *d_triangles = nullptr;
        hipMalloc((void **) &d_triangles, object.size * sizeof(Triangle));

        for (int j = 0; j < object.size; j++) {
            const Triangle triangle = object.triangles[j];

            hipMemcpy(&d_triangles[j], &triangle, sizeof(Triangle), hipMemcpyHostToDevice);
        }

        Object tmp_object = object;
        tmp_object.triangles = d_triangles;

        hipMemcpy(&d_objects[i], &tmp_object, sizeof(Object), hipMemcpyHostToDevice);
    }

    Landscape tmp_landscape = landscape;
    tmp_landscape.objects = d_objects;

    hipMemcpy(d_landscape, &tmp_landscape, sizeof(Landscape), hipMemcpyHostToDevice);
}

__host__ void freeLandscapeFromGPU(
    const Landscape landscape,
    Landscape *d_landscape
)
{
}

extern "C" void generateCoordinatesOnGPU(
    const int width,
    const int height,
    const int x,
    const double aspectRatio,
    Coordinates *output
)
{
    Coordinates *d_output = nullptr;
    const size_t size = width * height * sizeof(Coordinates);

    hipMalloc((void **) &d_output, size);

    generateCoordinatesKernel<<<1, height>>>(width, height, x, aspectRatio, d_output);
    hipDeviceSynchronize();

    const auto error = hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    if (error != hipSuccess) {
        printf("CUDA memcpy error in coordinate generation: %s\n", hipGetErrorString(error));
    }

    hipFree(d_output);
}

extern "C" void determineLandscapeIntersectionsOnGPU(
    const Line *lines,
    const Landscape landscape,
    const int width,
    const int height,
    LineTriangleIntersection *objectIntersections
)
{
    Landscape *d_landscape = nullptr;
    LineTriangleIntersection *d_output = nullptr;
    const size_t size = width * height * sizeof(LineTriangleIntersection);

    copyLandscapeToGPU(landscape, &d_landscape);
    hipError_t error = hipMalloc((void **) &d_output, size);

    if (error != hipSuccess) {
        printf("CUDA malloc error: %s\n", hipGetErrorString(error));
    }

    intersectionKernel<<<width, height>>>(lines, d_landscape, width, height, d_output);
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(error));
    }

    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        printf("CUDA memcpy error: %s\n", hipGetErrorString(error));
        std::exit(-1);
    }

    error = hipMemcpy(
        objectIntersections,
        d_output,
        size,
        hipMemcpyDeviceToHost
    );

    if (error != hipSuccess) {
        printf("CUDA memcpy error: %s\n", hipGetErrorString(error));
        // std::exit(-1);
    }

    hipFree(d_output);
}
