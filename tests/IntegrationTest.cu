#include <gtest/gtest.h>
#include <kernels.h>
#include <ViewPort.h>
#include "../include/Structs.h"

TEST(IntegrationTest, it_detects_intersection_with_rays_and_triangles)
{
    const int width = 1;
    const int height = 1;

    const auto viewport = ViewPort();
    const Line *lines = viewport.generateRays(width, height);
    constexpr int numberOfObjects = 1;
    Object objects[numberOfObjects] = {
        Object{
            1,
            new Triangle[1]{
                Triangle{
                    Vector{1.0f, 1.0f, 5.0f},
                    Vector{-1.0f, 1.0f, 5.0f},
                    Vector{0.0f, -1.0f, 5.0f}
                }
            }
        }
    };

    const Landscape landscape = {
        numberOfObjects,
        objects
    };

    auto *objectIntersections = new LineTriangleIntersection[width * height];
    determineLandscapeIntersections(
        lines,
        landscape,
        width,
        height,
        objectIntersections
    );

    ASSERT_TRUE(objectIntersections[0].intersects);
}

TEST(Integrationtest, it_copies_data_to_gpu)
{
    const int expectedSize = 1;

    struct Test {
        int size;
    };

    Test* d_test = nullptr;
    hipMalloc(&d_test, sizeof(Test));

    Test tmp_test{};
    tmp_test.size = expectedSize;

    Test *copyInto = new Test();

    hipMemcpy(d_test, &tmp_test, sizeof(Test), hipMemcpyHostToDevice);
    hipMemcpy(copyInto, d_test, sizeof(Test), hipMemcpyDeviceToHost);
    printf("Test size: %d\n", tmp_test.size);  // Should print 1

    ASSERT_EQ(copyInto->size, expectedSize);
}
